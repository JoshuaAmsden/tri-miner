#include "hip/hip_runtime.h"
/**
 * X16R algorithm (X16 with Randomized chain order)
 *
 * tpruvot 2018 - GPL code
 */

#include <stdio.h>
#include <memory.h>
#include <unistd.h>
#include <string.h>

extern "C" {
#include "sph/sph_blake.h"
#include "sph/sph_bmw.h"
#include "sph/sph_groestl.h"
#include "sph/sph_skein.h"
#include "sph/sph_jh.h"
#include "sph/sph_keccak.h"

#include "sph/sph_luffa.h"
#include "sph/sph_cubehash.h"
#include "sph/sph_shavite.h"
#include "sph/sph_simd.h"
#include "sph/sph_echo.h"

#include "sph/sph_hamsi.h"
#include "sph/sph_fugue.h"
#include "sph/sph_shabal.h"
#include "sph/sph_whirlpool.h"
#include "sph/sph_sha2.h"
#include "sph/sph_haval.h"
}

#include "uint256.h"
#include "miner.h"
#include "cuda_helper.h"
#include "cuda_x16.h"
//#include "hash_selection.h"

using namespace std;

static uint32_t *d_hash[MAX_GPUS];

enum Algo {
	BLAKE = 0,
	BMW,
	GROESTL,
	JH,
	KECCAK,
	SKEIN,
	LUFFA,
	CUBEHASH,
	SHAVITE,
	SIMD,
	ECHO,
	HAMSI,
	FUGUE,
	SHABAL,
	WHIRLPOOL,
	SHA512,
	HAVAL,
	HASH_FUNC_COUNT
};

static const char* algo_strings[] = {
	"blake",
	"bmw512",
	"groestl",
	"jh512",
	"keccak",
	"skein",
	"luffa",
	"cube",
	"shavite",
	"simd",
	"echo",
	"hamsi",
	"fugue",
	"shabal",
	"whirlpool",
	"sha512",
	"haval",
	NULL
};

static __thread uint32_t s_ntime = UINT32_MAX;
static __thread bool s_implemented = false;
static __thread char hashOrder[HASH_FUNC_COUNT + 1] = { 0 };

static bool isScrambleHash(const uint256& blockHash) {
	#define START_OF_LAST_35_NIBBLES_OF_HASH 29
	int last35Nibble = blockHash.GetNibble(START_OF_LAST_35_NIBBLES_OF_HASH);
	return (last35Nibble % 2 == 0);
}

static uint256 scrambleHash(const uint256& blockHash) {
	// Cliffnotes: use last 34 of PrevBlockHash to shuffle
	// a list of all algos and append that to PrevBlockHash and pass to hasher
	//////

	std::string hashString = blockHash.GetHex(); // uint256 to string
	std::string list = "0123456789abcdef";
	std::string order = list;
	std::string order2 = list;

	std::string hashFront = hashString.substr(0,30); // preserve first 30 chars
	std::string sixteen2 = hashString.substr(30,46); // extract last 19-34 chars
	std::string sixteen = hashString.substr(46,62); // extract last 3-18 chars
	std::string last2 = hashString.substr(62,64); // extract last 2 chars
	for(int i=0; i<16; i++){
	  int offset = list.find(sixteen[i]); // find offset of 16 char

	  order.insert(0, 1, order[offset]); // insert the nth character at the beginning
	  order.erase(offset+1, 1);  // erase the n+1 character (was nth)
	}

	for(int j=0; j<16; j++){
	  int offset = list.find(sixteen2[j]); // find offset of 16 char

	  order2.insert(0, 1, order2[offset]); // insert the nth character at the beginning
	  order2.erase(offset+1, 1);  // erase the n+1 character (was nth)
	}
	int offset = list.find(last2[0]); // find offset of 16 char
	order2.insert(0, 1, order2[offset]);
	offset = list.find(last2[1]); // find offset of 16 char
	order2.insert(0, 1, order2[offset]);
	uint256 scrambleHash = uint256(hashFront + order2 + order); // uint256 with length of hash and shuffled last seventeen
	return scrambleHash;
}

static uint8_t GetSelection(const uint256& blockHash, const int index) {
	//assert(index >= 0);
	///assert(index < 17);

	#define START_OF_LAST_17_NIBBLES_OF_HASH 47
	uint8_t hashSelection = blockHash.GetNibble(START_OF_LAST_17_NIBBLES_OF_HASH + index);
	#define START_OF_LAST_34_NIBBLES_OF_HASH 30
	uint8_t additionalSelection = blockHash.GetNibble(START_OF_LAST_34_NIBBLES_OF_HASH + index);
	hashSelection += (additionalSelection % 2);
	return(hashSelection);
}

/*
static void getAlgoScrambleString(const uint32_t* prevblock, char *output)
{
	uint8_t* data = (uint8_t*)prevblock;

	strcpy(output, "000123456789ABCDEF0123456789ABCDEF");

	for(int i = 2; i < 18; i++){
		uint8_t b = (17 - i) >> 1; // 16 ascii hex chars, reversed
		uint8_t algoDigit = (i & 1) ? data[b] & 0xF : data[b] >> 4;
		int offset = algoDigit + 2;
		// insert the nth character at the front
		char oldVal = output[offset];
		for(int j=offset; j-->0;)
			output[j+1] = output[j];
		output[2] = oldVal;
	}
	for(i < 34; i++){
		uint8_t b = (33 - i) >> 1; // 16 ascii hex chars, reversed
		uint8_t algoDigit = (i & 1) ? data[b] & 0xF : data[b] >> 4;
		int offset = algoDigit + 16;
		// insert the nth character at the front
		char oldVal = output[offset];
		for(int j=offset; j-->0;)
			output[j+1] = output[j];
		output[18] = oldVal;
	}
	uint8_t algoDigit = data[1] & 0F;
	output[0] = algoDigit;
	algoDigit = data[0] >> 4;
	output[1]; = algoDigit;
}*/

static void getAlgoString(const uint32_t* prevblock, char *output)
{

	uint256 prevHash;
	prevHash.setUint32t(prevblock);
	applog(LOG_NOTICE, "prevhash %s\n", prevHash.GetHex().c_str());
	bool toBeScamble = isScrambleHash(prevHash);
	uint256 hash;
	if(toBeScamble) {
		hash = scrambleHash(prevHash);
	} else {
		hash = prevHash;
	}
	char *sptr = output;
	printf("hash selection %s:",  hash.GetHex().c_str());
	for(int i = 0; i < 17; i ++) {
		uint8_t hashSelection =  GetSelection(hash, i);
		printf("%u,", hashSelection);
		if (hashSelection >= 10) {
			//printf("%c", 'A' + (hashSelection - 10));
			sprintf(sptr, "%c", 'A' + (hashSelection - 10));
		}
		else {
			//printf("%d", hashSelection);
			sprintf(sptr, "%u", (uint32_t) hashSelection);
		}
		sptr++;
	}
	*sptr = '\0';
	printf("----%s\n", output);
}

// Trihash CPU Hash (Validation)
extern "C" void trihash(void *output, const void *input)
{
	unsigned char _ALIGN(64) hash[136];

	sph_blake512_context ctx_blake;
	sph_bmw512_context ctx_bmw;
	sph_groestl512_context ctx_groestl;
	sph_jh512_context ctx_jh;
	sph_keccak512_context ctx_keccak;
	sph_skein512_context ctx_skein;
	sph_luffa512_context ctx_luffa;
	sph_cubehash512_context ctx_cubehash;
	sph_shavite512_context ctx_shavite;
	sph_simd512_context ctx_simd;
	sph_echo512_context ctx_echo;
	sph_hamsi512_context ctx_hamsi;
	sph_fugue512_context ctx_fugue;
	sph_shabal512_context ctx_shabal;
	sph_whirlpool_context ctx_whirlpool;
	sph_sha512_context ctx_sha512;
	sph_haval256_5_context   ctx_haval;

	void *in = (void*) input;
	int size = 80;

	uint32_t *in32 = (uint32_t*) input;
	getAlgoString(&in32[1], hashOrder);

	for (int i = 0; i < 17; i++)
	{
		const char elem = hashOrder[i];
		const uint8_t algo = elem >= 'A' ? elem - 'A' + 10 : elem - '0';

		switch (algo) {
		case BLAKE:
			sph_blake512_init(&ctx_blake);
			sph_blake512(&ctx_blake, in, size);
			sph_blake512_close(&ctx_blake, hash);
			break;
		case BMW:
			sph_bmw512_init(&ctx_bmw);
			sph_bmw512(&ctx_bmw, in, size);
			sph_bmw512_close(&ctx_bmw, hash);
			break;
		case GROESTL:
			sph_groestl512_init(&ctx_groestl);
			sph_groestl512(&ctx_groestl, in, size);
			sph_groestl512_close(&ctx_groestl, hash);
			break;
		case SKEIN:
			sph_skein512_init(&ctx_skein);
			sph_skein512(&ctx_skein, in, size);
			sph_skein512_close(&ctx_skein, hash);
			break;
		case JH:
			sph_jh512_init(&ctx_jh);
			sph_jh512(&ctx_jh, in, size);
			sph_jh512_close(&ctx_jh, hash);
			break;
		case KECCAK:
			sph_keccak512_init(&ctx_keccak);
			sph_keccak512(&ctx_keccak, in, size);
			sph_keccak512_close(&ctx_keccak, hash);
			break;
		case LUFFA:
			sph_luffa512_init(&ctx_luffa);
			sph_luffa512(&ctx_luffa, in, size);
			sph_luffa512_close(&ctx_luffa, hash);
			break;
		case CUBEHASH:
			sph_cubehash512_init(&ctx_cubehash);
			sph_cubehash512(&ctx_cubehash, in, size);
			sph_cubehash512_close(&ctx_cubehash, hash);
			break;
		case SHAVITE:
			sph_shavite512_init(&ctx_shavite);
			sph_shavite512(&ctx_shavite, in, size);
			sph_shavite512_close(&ctx_shavite, hash);
			break;
		case SIMD:
			sph_simd512_init(&ctx_simd);
			sph_simd512(&ctx_simd, in, size);
			sph_simd512_close(&ctx_simd, hash);
			break;
		case ECHO:
			sph_echo512_init(&ctx_echo);
			sph_echo512(&ctx_echo, in, size);
			sph_echo512_close(&ctx_echo, hash);
			break;
		case HAMSI:
			sph_hamsi512_init(&ctx_hamsi);
			sph_hamsi512(&ctx_hamsi, in, size);
			sph_hamsi512_close(&ctx_hamsi, hash);
			break;
		case FUGUE:
			sph_fugue512_init(&ctx_fugue);
			sph_fugue512(&ctx_fugue, in, size);
			sph_fugue512_close(&ctx_fugue, hash);
			break;
		case SHABAL:
			sph_shabal512_init(&ctx_shabal);
			sph_shabal512(&ctx_shabal, in, size);
			sph_shabal512_close(&ctx_shabal, hash);
			break;
		case WHIRLPOOL:
			sph_whirlpool_init(&ctx_whirlpool);
			sph_whirlpool(&ctx_whirlpool, in, size);
			sph_whirlpool_close(&ctx_whirlpool, hash);
			break;
		case SHA512:
			sph_sha512_init(&ctx_sha512);
			sph_sha512(&ctx_sha512,(const void*) in, size);
			sph_sha512_close(&ctx_sha512,(void*) hash);
			break;
		case HAVAL:
			printf("hashing havals\n");
			sph_haval256_5_init(&ctx_haval);
			sph_haval256_5(&ctx_haval,(const void*) in, size);
			sph_haval256_5_close(&ctx_haval,hash);
		   break;
		}
		in = (void*) hash;
		size = 64;
	}
	memcpy(output, hash, 32);
}

static bool init[MAX_GPUS] = { 0 };

//#define _DEBUG
#define _DEBUG_PREFIX "trihash-"
#include "cuda_debug.cuh"

// #define GPU_HASH_CHECK_LOG

#ifdef GPU_HASH_CHECK_LOG
	static int algo80_tests[HASH_FUNC_COUNT] = { 0 };
	static int algo64_tests[HASH_FUNC_COUNT] = { 0 };
#endif
static int algo80_fails[HASH_FUNC_COUNT] = { 0 };

extern "C" int scanhash_trihash(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t first_nonce = pdata[19];
	const int dev_id = device_map[thr_id];
	int intensity = (device_sm[dev_id] > 500 && !is_windows()) ? 20 : 19;
	if (strstr(device_name[dev_id], "GTX 1080")) intensity = 19;
	uint32_t throughput = cuda_default_throughput(thr_id, 1U << intensity);

	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
		}
		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);

		//quark_blake512_cpu_init(thr_id, throughput); // Redundant
		//quark_bmw512_cpu_init(thr_id, throughput); // Redundant
		quark_groestl512_cpu_init(thr_id, throughput);
		//quark_skein512_cpu_init(thr_id, throughput); // Redundant
		//quark_jh512_cpu_init(thr_id, throughput); // Redundant
		quark_keccak512_cpu_init(thr_id, throughput);
		x11_shavite512_cpu_init(thr_id, throughput);
		x11_simd512_cpu_init(thr_id, throughput); // 64
		x13_hamsi512_cpu_init(thr_id, throughput);
		x16_fugue512_cpu_init(thr_id, throughput);
		x15_whirlpool_cpu_init(thr_id, throughput, 0);
		x16_whirlpool512_init(thr_id, throughput);
		x17_sha512_cpu_init(thr_id, throughput);
		x17_haval256_cpu_init(thr_id, throughput);

		CUDA_CALL_OR_RET_X(hipMalloc(&d_hash[thr_id], (size_t) 64 * throughput), 0);

		cuda_check_cpu_init(thr_id, throughput);

		init[thr_id] = true;
	}

	if (opt_benchmark) {
		((uint32_t*)ptarget)[7] = 0x003f;
		((uint32_t*)pdata)[1] = 0xEFCDAB89;
		((uint32_t*)pdata)[2] = 0x67452301;
	}
	uint32_t _ALIGN(64) endiandata[20];

	for (int k=0; k < 19; k++)
		be32enc(&endiandata[k], pdata[k]);

	uint32_t ntime = swab32(pdata[17]);
	if (s_ntime != ntime) {
		getAlgoString(&endiandata[1], hashOrder);
		s_ntime = ntime;
		s_implemented = true;
		if (!thr_id) applog(LOG_INFO, "hash order1 %s (%08x)\n", hashOrder, ntime);
	}

	if (!s_implemented) {
	 	applog(LOG_INFO, "s_implemented is false, wait 1 min to terminate %s \n", hashOrder);
		sleep(1);
		return -1;
	}

	cuda_check_cpu_setTarget(ptarget);

	char elem = hashOrder[0];
	const uint8_t algo80 = elem >= 'A' ? elem - 'A' + 10 : elem - '0';

	switch (algo80) {
		case BLAKE:
			quark_blake512_cpu_setBlock_80(thr_id, endiandata);
			break;
		case BMW:
			quark_bmw512_cpu_setBlock_80(endiandata);
			break;
		case GROESTL:
			groestl512_setBlock_80(thr_id, endiandata);
			break;
		case JH:
			jh512_setBlock_80(thr_id, endiandata);
			break;
		case KECCAK:
			keccak512_setBlock_80(thr_id, endiandata);
			break;
		case SKEIN:
			skein512_cpu_setBlock_80((void*)endiandata);
			break;
		case LUFFA:
			qubit_luffa512_cpu_setBlock_80_alexis((void*)endiandata);
			break;
		case CUBEHASH:
			cubehash512_setBlock_80(thr_id, endiandata);
			break;
		case SHAVITE:
			x11_shavite512_setBlock_80((void*)endiandata);
			break;
		case SIMD:
			x16_simd512_setBlock_80((void*)endiandata);
			break;
		case ECHO:
			x11_echo512_setBlock_80_alexis((void*)endiandata);
			break;
		case HAMSI:
			x16_hamsi512_setBlock_80((void*)endiandata);
			break;
		case FUGUE:
			x16_fugue512_setBlock_80((void*)pdata);
			break;
		case SHABAL:
			x16_shabal512_setBlock_80((void*)endiandata);
			break;
		case WHIRLPOOL:
			x16_whirlpool512_setBlock_80((void*)endiandata);
			break;
		case SHA512:
			x16_sha512_setBlock_80(endiandata);
			break;
		case HAVAL: //TODO: implement setblock_80 for haval256
			//x17_haval256_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id], 256); order++;
			break;
		default: {
			if (!thr_id)
				applog(LOG_WARNING, "kernel %s %c unimplemented, order %s", algo_strings[algo80], elem, hashOrder);
			s_implemented = false;
			applog(LOG_INFO, "s_implemented is false, wait 5 min to terminate %s \n", hashOrder);
			sleep(5);
			return -1;
		}
	}

	int warn = 0;

	do {
		int order = 0;

		// Hash with CUDA

		switch (algo80) {
			case BLAKE:
				quark_blake512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id]); order++;
				TRACE("blake80:");
				break;
			case BMW:
				quark_bmw512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
				TRACE("bmw80  :");
				break;
			case GROESTL:
				groestl512_cuda_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id]); order++;
				TRACE("grstl80:");
				break;
			case JH:
				jh512_cuda_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id]); order++;
				TRACE("jh51280:");
				break;
			case KECCAK:
				keccak512_cuda_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id]); order++;
				TRACE("kecck80:");
				break;
			case SKEIN:
				skein512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id], 1); order++;
				TRACE("skein80:");
				break;
			case LUFFA:
				qubit_luffa512_cpu_hash_80_alexis(thr_id, throughput, pdata[19], d_hash[thr_id]); order++;
				TRACE("luffa80:");
				break;
			case CUBEHASH:
				cubehash512_cuda_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id]); order++;
				TRACE("cube 80:");
				break;
			case SHAVITE:
				x11_shavite512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
				TRACE("shavite:");
				break;
			case SIMD:
				x16_simd512_cuda_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id]); order++;
				TRACE("simd512:");
				break;
			case ECHO:
				x11_echo512_cpu_hash_80_alexis(thr_id, throughput, pdata[19], d_hash[thr_id]); order++;
				TRACE("echo   :");
				break;
			case HAMSI:
				x16_hamsi512_cuda_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id]); order++;
				TRACE("hamsi  :");
				break;
			case FUGUE:
				x16_fugue512_cuda_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id]); order++;
				TRACE("fugue  :");
				break;
			case SHABAL:
				x16_shabal512_cuda_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id]); order++;
				TRACE("shabal :");
				break;
			case WHIRLPOOL:
				x16_whirlpool512_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id]); order++;
				TRACE("whirl  :");
				break;
			case SHA512:
				x16_sha512_cuda_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id]); order++;
				TRACE("sha512 :");
				break;
			case HAVAL:
				order++;
				//x17_haval256_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id], 256); order++;
				TRACE("haval256");
				break;
		}

		for (int i = 1; i < 17; i++)
		{
			const char elem = hashOrder[i];
			const uint8_t algo64 = elem >= 'A' ? elem - 'A' + 10 : elem - '0';

			switch (algo64) {
			case BLAKE:
				quark_blake512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
				TRACE("blake  :");
				break;
			case BMW:
				quark_bmw512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
				TRACE("bmw    :");
				break;
			case GROESTL:
				quark_groestl512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
				TRACE("groestl:");
				break;
			case JH:
				quark_jh512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
				TRACE("jh512  :");
				break;
			case KECCAK:
				quark_keccak512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
				TRACE("keccak :");
				break;
			case SKEIN:
				quark_skein512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
				TRACE("skein  :");
				break;
			case LUFFA:
				x11_luffa512_cpu_hash_64_alexis(thr_id, throughput, d_hash[thr_id]); order++;
				TRACE("luffa  :");
				break;
			case CUBEHASH:
				x11_cubehash512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
				TRACE("cube   :");
				break;
			case SHAVITE:
				x11_shavite512_cpu_hash_64_alexis(thr_id, throughput, d_hash[thr_id]); order++;
				TRACE("shavite:");
				break;
			case SIMD:
				x11_simd512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
				TRACE("simd   :");
				break;
			case ECHO:
				x11_echo512_cpu_hash_64_alexis(thr_id, throughput, d_hash[thr_id]); order++;
				TRACE("echo   :");
				break;
			case HAMSI:
				x13_hamsi512_cpu_hash_64_alexis(thr_id, throughput, d_hash[thr_id]); order++;
				TRACE("hamsi  :");
				break;
			case FUGUE:
				x13_fugue512_cpu_hash_64_alexis(thr_id, throughput, d_hash[thr_id]); order++;
				TRACE("fugue  :");
				break;
			case SHABAL:
				x14_shabal512_cpu_hash_64_alexis(thr_id, throughput, d_hash[thr_id]); order++;
				TRACE("shabal :");
				break;
			case WHIRLPOOL:
				x15_whirlpool_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
				TRACE("shabal :");
				break;
			case SHA512:
				x17_sha512_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id]); order++;
				TRACE("sha512 :");
				break;
			case HAVAL:
				x17_haval256_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id], 256); order++;
				TRACE("haval256");
				break;
			}
		}

		*hashes_done = pdata[19] - first_nonce + throughput;

		work->nonces[0] = cuda_check_hash(thr_id, throughput, pdata[19], d_hash[thr_id]);
#ifdef _DEBUG
		uint32_t _ALIGN(64) dhash[8];
		be32enc(&endiandata[19], pdata[19]);
		trihash(dhash, endiandata);
		applog_hash(dhash);
		return -1;
#endif
		if (work->nonces[0] != UINT32_MAX)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t _ALIGN(64) vhash[8];
			be32enc(&endiandata[19], work->nonces[0]);
			trihash(vhash, endiandata);

			if (vhash[7] <= Htarg && fulltest(vhash, ptarget)) {
				work->valid_nonces = 1;
				work->nonces[1] = cuda_check_hash_suppl(thr_id, throughput, pdata[19], d_hash[thr_id], 1);
				work_set_target_ratio(work, vhash);
				if (work->nonces[1] != 0) {
					be32enc(&endiandata[19], work->nonces[1]);
					trihash(vhash, endiandata);
					bn_set_target_ratio(work, vhash, 1);
					work->valid_nonces++;
					pdata[19] = max(work->nonces[0], work->nonces[1]) + 1;
				} else {
					pdata[19] = work->nonces[0] + 1; // cursor
				}
				#ifdef GPU_HASH_CHECK_LOG
				gpulog(LOG_INFO, thr_id, "hash found with %s 80!", algo_strings[algo80]);

				algo80_tests[algo80] += work->valid_nonces;
				char oks64[128] = { 0 };
				char oks80[128] = { 0 };
				char fails[128] = { 0 };
				for (int a = 0; a < HASH_FUNC_COUNT; a++) {
						const char elem = hashOrder[a];
						const uint8_t algo64 = elem >= 'A' ? elem - 'A' + 10 : elem - '0';
						if (a > 0) algo64_tests[algo64] += work->valid_nonces;
						sprintf(&oks64[strlen(oks64)], "|%X:%2d", a, algo64_tests[a] < 100 ? algo64_tests[a] : 99);
						sprintf(&oks80[strlen(oks80)], "|%X:%2d", a, algo80_tests[a] < 100 ? algo80_tests[a] : 99);
						sprintf(&fails[strlen(fails)], "|%X:%2d", a, algo80_fails[a] < 100 ? algo80_fails[a] : 99);
				}
				applog(LOG_INFO, "K64: %s", oks64);
				applog(LOG_INFO, "K80: %s", oks80);
				applog(LOG_ERR,  "F80: %s", fails);
				#endif
				return work->valid_nonces;
			}
			else if (vhash[7] > Htarg) {
				// x11+ coins could do some random error, but not on retry
				gpu_increment_reject(thr_id);
				algo80_fails[algo80]++;
				if (!warn) {
					warn++;
					pdata[19] = work->nonces[0] + 1;
					continue;
				} else {
					if (!opt_quiet)	gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU! %s %s",
						work->nonces[0], algo_strings[algo80], hashOrder);
					warn = 0;
				}
			}
		}

		if ((uint64_t)throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}

		pdata[19] += throughput;

	} while (pdata[19] < max_nonce && !work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce;
	return 0;
}

// cleanup
extern "C" void free_trihash(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	hipFree(d_hash[thr_id]);

	quark_blake512_cpu_free(thr_id);
	quark_groestl512_cpu_free(thr_id);
	x11_simd512_cpu_free(thr_id);
	x16_fugue512_cpu_free(thr_id); // to merge with x13_fugue512 ?
	x15_whirlpool_cpu_free(thr_id);

	cuda_check_cpu_free(thr_id);

	hipDeviceSynchronize();
	init[thr_id] = false;
}
